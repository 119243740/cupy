#include <stdio.h>
#include "cupy_cutensor.h"

void _cutensor_alloc_handle(hiptensorHandle_t **handle)
{
    *handle = (hiptensorHandle_t*) malloc( sizeof(hiptensorHandle_t) );
}

void _cutensor_free_handle(hiptensorHandle_t *handle)
{
    free(handle);
}

void _cutensor_alloc_tensor_descriptor(hiptensorTensorDescriptor_t **desc)
{
    *desc = (hiptensorTensorDescriptor_t*) malloc( sizeof(hiptensorTensorDescriptor_t) );
}

void _cutensor_free_tensor_descriptor(hiptensorTensorDescriptor_t *desc)
{
    free(desc);
}

void _cutensor_alloc_contraction_descriptor(cutensorContractionDescriptor_t **desc)
{
    *desc = (cutensorContractionDescriptor_t*) malloc( sizeof(cutensorContractionDescriptor_t) );
}

void _cutensor_free_contraction_descriptor(cutensorContractionDescriptor_t *desc)
{
    free(desc);
}

void _cutensor_alloc_contraction_plan(hiptensorContractionPlan_t **plan)
{
    *plan = (hiptensorContractionPlan_t*) malloc( sizeof(hiptensorContractionPlan_t) );
}

void _cutensor_free_contraction_plan(hiptensorContractionPlan_t *plan)
{
    free(plan);
}

void _cutensor_alloc_contraction_find(cutensorContractionFind_t **find)
{
    *find = (cutensorContractionFind_t*) malloc( sizeof(cutensorContractionFind_t) );
}

void _cutensor_free_contraction_find(cutensorContractionFind_t *find)
{
    free(find);
}
